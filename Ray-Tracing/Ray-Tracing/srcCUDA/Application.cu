#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#include "utils/Cuda.h"
#include "utils/Vec3.h"
#include "utils/Utils.h"
#include "utils/Ray.h"

#include "objects/objects/Sphere.h"
#include "objects/objects/MovingSphere.h"
#include "objects/objects/HittableList.h"
#include "objects/materials/Material.h"
#include "objects/RayTracer.h"

#include "Camera.h"
#include "Renderer.h"

//#include "Scene.h"

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(6211, 0, 0, rand_state);
    }
}

__global__ void BuildScene1(Hittable** d_list, Hittable** d_world, Camera** d_Camera, int image_width, int image_height, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        d_list[0] = new Sphere(Point3(0, -1000.0, -1), 1000, new Lambertian(&CheckerTexture(Color(1.0, 0.0, 1.0), Color(0.0, 0.0, 0.0))));

        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                Point3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    Point3 center2 = center + Vec3(0, 0.5f * RND, 0);
                    d_list[i++] = new MovingSphere(center, center2, 0.0f, 1.0f, 0.2f, new Lambertian(Color(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new Sphere(center, 0.2f, new Metal(Color(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = new Sphere(center, 0.2f, new Dielectric(1.5));
                }
            }
        }

        d_list[i++] = new Sphere(Point3(0, 1, 0), 1.0, new Dielectric(1.5));
        d_list[i++] = new Sphere(Point3(-4, 1, 0), 1.0, new Lambertian(Color(0.4, 0.2, 0.1)));
        d_list[i++] = new Sphere(Point3(4, 1, 0), 1.0, new Metal(Color(0.7, 0.6, 0.5), 0.0));
        
        *rand_state = local_rand_state;
        *d_world = new HittableList(d_list, 22 * 22 + 1 + 3);

        Point3 lookfrom(13, 2, 3);
        Point3 lookat(0, 0, 0);
        float dist_to_focus = (lookfrom - lookat).Length();
        float aperture = 0.1;
        *d_Camera = new Camera(lookfrom,
            lookat,
            Vec3(0, 1, 0),
            30.0,
            float(image_width) / float(image_height),
            aperture,
            dist_to_focus,
            0.0f,
            1.0f);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) 
        return;
    int pixel_index = j * max_x + i;
    // Each thread gets different seed
    hiprand_init(6211 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(Color *frame_buffer, int max_x, int max_y, int samples_per_pixel, Camera **cam, Hittable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) 
        return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    Color color(0, 0, 0);
    for(int s = 0; s < samples_per_pixel; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        Ray r = (*cam)->GetRay(u, v, &local_rand_state);
        color += RayColor(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    color /= float(samples_per_pixel);
    color[0] = sqrt(color[0]);
    color[1] = sqrt(color[1]);
    color[2] = sqrt(color[2]);
    frame_buffer[pixel_index] = color;
}

__global__ void free_world(Hittable **d_list, Hittable **d_world, Camera **d_Camera) {
    for(int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((Sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_Camera;
}

int main() {
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    printf("Device ID: %d \t Number of SMs: %d\n", deviceId, numberOfSMs);

    int image_width = 400;
    int image_height = 300;
    int samples_per_pixel = 10;
    int block_width = 16;
    int block_height = 16;

    std::cout << "Rendering a " << image_width << "x" << image_height << " image with " << samples_per_pixel << " samples per pixel ";
    std::cout << "in " << block_width << "x" << block_height << " blocks.\n";

    // allocate frame buffer
    int num_pixels = image_width * image_height;
    size_t frame_buffer_size = num_pixels * sizeof(Color);
    Color* frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random states
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // build the world of hittables & the camera
    Hittable **d_list;
    int num_Hittables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_Hittables * sizeof(Hittable *)));

    Hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hittable *)));
    
    Camera **d_Camera;
    checkCudaErrors(hipMalloc((void **)&d_Camera, sizeof(Camera *)));
    
    std::cout << "Building scene \n";
    BuildScene1<<<1, 1>>>(d_list, d_world, d_Camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();

    // Render the frame buffer
    dim3 blocks(image_width / block_width + 1, image_height / block_height + 1);
    dim3 threads(block_width, block_height);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "Rendering image to frame buffer\n";
    render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_Camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // get total time
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cout << "took " << timer_seconds << " seconds.\n";

    // Output frame_buffer as Image
    Renderer renderer(image_width, image_height);
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            int ir = int(255.99*frame_buffer[pixel_index].x());
            int ig = int(255.99*frame_buffer[pixel_index].y());
            int ib = int(255.99*frame_buffer[pixel_index].z());
            renderer.WriteColors(ir, ig, ib);
        }
    }
    std::cout << "Writing output file \n";
    renderer.WriteOutput(image_width, image_height);
    std::cout << "Done.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_Camera);
    checkCudaErrors(hipGetLastError());

    // free memory
    checkCudaErrors(hipFree(d_Camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

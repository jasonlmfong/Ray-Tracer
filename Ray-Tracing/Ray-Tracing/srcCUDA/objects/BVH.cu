#include "BVH.h"

__device__ BVHNode::BVHNode()
{
}

__device__ BVHNode::BVHNode(const HittableList& hitList, float time0, float time1, hiprandState* local_rand_state)
    : BVHNode(hitList.list, 0, hitList.list_size * sizeof(Hittable), time0, time1, local_rand_state)
{
}

__device__ BVHNode::BVHNode(const thrust::device_vector<Hittable*>& src_objects, size_t start, size_t end, float time0, float time1, hiprandState* local_rand_state)
{
    auto objects = src_objects; // Create a modifiable array of the source scene objects

    int axis = RandomInt(0, 2, local_rand_state);
    auto comparator = (axis == 0) ? BoxXCompare : (axis == 1) ? BoxYCompare : BoxZCompare;

    size_t object_span = end - start;

    if (object_span == 1) {
        left = right = objects[start];
    }
    else if (object_span == 2) {
        if (comparator(objects[start], objects[start + 1])) {
            left = objects[start];
            right = objects[start + 1];
        }
        else {
            left = objects[start + 1];
            right = objects[start];
        }
    }
    else {
        std::sort(objects.begin() + start, objects.begin() + end, comparator);

        auto mid = start + object_span / 2;
        left = &BVHNode(objects, start, mid, time0, time1, local_rand_state);
        right = &BVHNode(objects, mid, end, time0, time1, local_rand_state);
    }

    AABB box_left, box_right;

    if (!left->BoundingBox(time0, time1, box_left)
        || !right->BoundingBox(time0, time1, box_right)
        )
        printf("No bounding box in bvh_node constructor.\n");

    box = SurroundingBox(box_left, box_right);
}

__device__ bool BVHNode::Hit(const Ray& r, float t_min, float t_max, hitRecord& rec) const
{
    if (!box.Hit(r, t_min, t_max))
        return false;

    bool hit_left = left->Hit(r, t_min, t_max, rec);
    bool hit_right = right->Hit(r, t_min, hit_left ? rec.t : t_max, rec);

    return hit_left || hit_right;
}

__device__ bool BVHNode::BoundingBox(float time0, float time1, AABB& output_box) const
{
    output_box = box;
    return true;
}

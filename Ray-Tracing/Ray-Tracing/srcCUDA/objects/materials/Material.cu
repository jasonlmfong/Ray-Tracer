#include "hip/hip_runtime.h"
#include "Material.h"

__device__ bool Material::Scatter(const Ray& r_in, const hitRecord& rec, Vec3& attenuation, Ray& Scattered, hiprandState* local_rand_state) const
{
    return false;
}

__device__ Lambertian::Lambertian(const Color& a)
    : albedo(&SolidColor(a))
{
}

__device__ Lambertian::Lambertian(Texture* a)
    : albedo(a)
{
}

__device__ bool Lambertian::Scatter(const Ray& r_in, const hitRecord& rec, Vec3& attenuation, Ray& Scattered, hiprandState* local_rand_state) const
{
    Vec3 scatter_direction = rec.normal + RandomUnitVector(local_rand_state);

    // Catch degenerate scatter direction
    if (scatter_direction.NearZero())
        scatter_direction = rec.normal;

    Scattered = Ray(rec.p, scatter_direction, r_in.GetTime());
    attenuation = albedo->GetValue(rec.u, rec.v, rec.p);
    return true;
}

__device__ Metal::Metal(const Vec3& a, float f)
    : albedo(a) 
{
    if (f < 1) fuzz = f; else fuzz = 1;
}

__device__ bool Metal::Scatter(const Ray& r_in, const hitRecord& rec, Vec3& attenuation, Ray& Scattered, hiprandState* local_rand_state) const
{
    Vec3 reflected = Reflect(UnitVector(r_in.GetDirection()), rec.normal);
    Scattered = Ray(rec.p, reflected + fuzz * RandomUnitVector(local_rand_state), r_in.GetTime());
    attenuation = albedo;
    return (Dot(Scattered.GetDirection(), rec.normal) > 0.0f);
}

__device__ Dielectric::Dielectric(float ri)
    : indexOfRefract(ri)
{
}

__device__ bool Dielectric::Scatter(const Ray& r_in, const hitRecord& rec, Vec3& attenuation, Ray& scattered, hiprandState* local_rand_state) const
{
    attenuation = Color(1.0f, 1.0f, 1.0f);
    float refraction_ratio = rec.front_face ? (1.0f / indexOfRefract) : indexOfRefract;

    Vec3 unit_direction = UnitVector(r_in.GetDirection());
    float cos_theta = fmin(Dot(-unit_direction, rec.normal), 1.0f);
    float sin_theta = sqrt(1.0f - cos_theta * cos_theta);

    bool cannot_refract = refraction_ratio * sin_theta > 1.0f;
    Vec3 direction;

    if (cannot_refract || Reflectance(cos_theta, refraction_ratio) > RandomDouble(local_rand_state))
        direction = Reflect(unit_direction, rec.normal);
    else
        direction = Refract(unit_direction, rec.normal, refraction_ratio);

    scattered = Ray(rec.p, direction, r_in.GetTime());
    return true;
}

__device__ float Dielectric::Reflectance(float cosine, float indexOfRefract) const
{
    float r0 = (1.0f - indexOfRefract) / (1.0f + indexOfRefract);
    r0 = r0 * r0;
    return r0 + (1.0f - r0) * pow((1.0f - cosine), 5.0f);
}

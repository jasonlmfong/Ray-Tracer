#include "hip/hip_runtime.h"
#include "Texture.h"

__device__ SolidColor::SolidColor()
{
}

__device__ SolidColor::SolidColor(Color c)
    : colorValue(c)
{
}

__device__ SolidColor::SolidColor(float red, float green, float blue)
    : SolidColor(Color(red, green, blue))
{
}

__device__ Color SolidColor::GetValue(float u, float v, const Vec3& p) const
{
    return colorValue;
}

__device__ CheckerTexture::CheckerTexture()
{
}

__device__ CheckerTexture::CheckerTexture(Texture* _even, Texture* _odd)
    : even(_even), odd(_odd)
{
}

__device__ CheckerTexture::CheckerTexture(Color c1, Color c2)
    : even(&SolidColor(c1)), odd(&SolidColor(c2))
{
}

__device__ Color CheckerTexture::GetValue(float u, float v, const Point3& p) const
{
    auto sines = sin(10 * p.x()) * sin(10 * p.y()) * sin(10 * p.z());
    if (sines < 0)
        return odd->GetValue(u, v, p);
    else
        return even->GetValue(u, v, p);
}

//__device__ NoiseTexture::NoiseTexture()
//{
//}
//
//__device__ Color NoiseTexture::GetValue(float u, float v, const Point3& p) const
//{
//    return Color(1, 1, 1) * m_Noise.Noise(p);
//}
//
//__device__ SmoothNoiseTexture::SmoothNoiseTexture()
//{
//}
//
//__device__ SmoothNoiseTexture::SmoothNoiseTexture(float sc)
//    : m_Scale(sc)
//{
//}
//
//__device__ Color SmoothNoiseTexture::GetValue(float u, float v, const Point3& p) const
//{
//    return Color(1, 1, 1) * 0.5 * (1 + sin(m_Scale * p.z() + 10 * m_Noise.Turbulence(p)));
//}
//
//__device__ ImageTexture::ImageTexture()
//    : m_Data(nullptr), m_Width(0), m_Height(0), bytesPerScanline(0)
//{
//}

//__device__ ImageTexture::ImageTexture(const char* filename)
//{
//    auto components_per_pixel = bytesPerPixel;
//
//    m_Data = stbi_load(
//        filename, &m_Width, &m_Height, &components_per_pixel, components_per_pixel);
//
//    if (!m_Data) {
//        std::cerr << "ERROR: Could not load texture image file '" << filename << "'.\n";
//        m_Width = m_Height = 0;
//    }
//
//    bytesPerScanline = bytesPerPixel * m_Width;
//}
//
//__device__ ImageTexture::~ImageTexture()
//{
//    delete m_Data;
//}
//
//__device__ Color ImageTexture::GetValue(float u, float v, const Vec3& p) const
//{
//    // If we have no texture data, then return solid cyan as a debugging aid.
//    if (m_Data == nullptr)
//        return Color(0, 1, 1);
//
//    // Clamp input texture coordinates to [0,1] x [1,0]
//    u = clamp(u, 0.0, 1.0);
//    v = 1.0 - clamp(v, 0.0, 1.0);  // Flip V to image coordinates
//
//    auto i = static_cast<int>(u * m_Width);
//    auto j = static_cast<int>(v * m_Height);
//
//    // Clamp integer mapping, since actual coordinates should be less than 1.0
//    if (i >= m_Width)  i = m_Width - 1;
//    if (j >= m_Height) j = m_Height - 1;
//
//    const auto color_scale = 1.0 / 255.0;
//    auto pixel = m_Data + j * bytesPerScanline + i * bytesPerPixel; // pointer arithmetic
//
//    return Color(color_scale * pixel[0], color_scale * pixel[1], color_scale * pixel[2]);
//}

#include "hip/hip_runtime.h"
#include "Sphere.h"

__device__ Sphere::Sphere()
{
}

__device__ Sphere::Sphere(Point3 cen, float r, Material* m)
	: center(cen), radius(r), mat_ptr(m)
{
}

__device__ bool Sphere::Hit(const Ray& r, float t_min, float t_max, hitRecord& rec) const {
    Vec3 oc = r.GetOrigin() - center;
    auto a = r.GetDirection().LengthSquared();
    auto half_b = Dot(oc, r.GetDirection());
    auto c = oc.LengthSquared() - radius * radius;

    auto discriminant = half_b * half_b - a * c;
    if (discriminant < 0) return false;
    auto sqrtd = sqrt(discriminant);

    // Find the nearest root that lies in the acceptable range.
    auto root = (-half_b - sqrtd) / a;
    if (root < t_min || t_max < root) {
        root = (-half_b + sqrtd) / a;
        if (root < t_min || t_max < root)
            return false;
    }

    rec.t = root;
    rec.p = r.GetPointAtTime(rec.t);
    Vec3 outward_normal = (rec.p - center) / radius;
    rec.SetFaceNormal(r, outward_normal);
    GetSphereUV(outward_normal, rec.u, rec.v);
    rec.mat_ptr = mat_ptr;

    return true;
}

//__device__ bool Sphere::BoundingBox(float time0, float time1, AABB& output_box) const
//{
//    output_box = AABB(
//        center - Vec3(radius, radius, radius),
//        center + Vec3(radius, radius, radius));
//    return true;
//}

__device__ void Sphere::GetSphereUV(const Point3& p, float& u, float& v)
{
    // p: a given point on the sphere of radius one, centered at the origin.
    // u: returned value [0,1] of angle around the Y axis from X=-1.
    // v: returned value [0,1] of angle from Y=-1 to Y=+1.
    //     <1 0 0> yields <0.50 0.50>       <-1  0  0> yields <0.00 0.50>
    //     <0 1 0> yields <0.50 1.00>       < 0 -1  0> yields <0.50 0.00>
    //     <0 0 1> yields <0.25 0.50>       < 0  0 -1> yields <0.75 0.50>

    auto theta = acos(-p.y());
    auto phi = atan2(-p.z(), p.x()) + PI;

    u = phi / (2 * PI);
    v = theta / PI;
}

#include "hip/hip_runtime.h"
#include "Vec3.h"

__host__ __device__ Vec3::Vec3()
    : e{ 0, 0, 0 }
{
}

__host__ __device__ Vec3::Vec3(float e0, float e1, float e2)
    : e{ e0, e1, e2 }
{
}

__host__ __device__ float Vec3::x() const
{
    return e[0];
}

__host__ __device__ float Vec3::y() const
{
    return e[1];
}

__host__ __device__ float Vec3::z() const
{
    return e[2];
}

__host__ __device__ Vec3 Vec3::operator-() const
{
    return Vec3(-e[0], -e[1], -e[2]);
}

__host__ __device__ float Vec3::operator[](int i) const
{
    return e[i];
}

__host__ __device__ float& Vec3::operator[](int i) {
    return e[i];
}

__host__ __device__ Vec3& Vec3::operator+=(const Vec3& v2)
{
    e[0] += v2.e[0];
    e[1] += v2.e[1];
    e[2] += v2.e[2];
    return *this;
}

__host__ __device__ Vec3& Vec3::operator*=(const Vec3& v2)
{
    e[0] *= v2.e[0];
    e[1] *= v2.e[1];
    e[2] *= v2.e[2];
    return *this;
}

__host__ __device__ Vec3& Vec3::operator/=(const Vec3& v2)
{
    e[0] /= v2.e[0];
    e[1] /= v2.e[1];
    e[2] /= v2.e[2];
    return *this;
}

__host__ __device__ Vec3& Vec3::operator-=(const Vec3& v2)
{
    e[0] -= v2.e[0];
    e[1] -= v2.e[1];
    e[2] -= v2.e[2];
    return *this;
}

__host__ __device__ Vec3& Vec3::operator*=(const float t)
{
    e[0] *= t;
    e[1] *= t;
    e[2] *= t;
    return *this;
}

__host__ __device__ Vec3& Vec3::operator/=(const float t)
{
    float k = 1.0 / t;

    e[0] *= k;
    e[1] *= k;
    e[2] *= k;
    return *this;
}

__host__ __device__ float Vec3::Length() const 
{ 
    return sqrt(e[0] * e[0] + e[1] * e[1] + e[2] * e[2]); 
}

__host__ __device__ float Vec3::LengthSquared() const
{ 
    return e[0] * e[0] + e[1] * e[1] + e[2] * e[2]; 
}

__host__ __device__ bool Vec3::NearZero() const
{
    // Return true if the vector is close to zero in all dimensions.
    const auto s = 1e-8;
    return (fabs(e[0]) < s) && (fabs(e[1]) < s) && (fabs(e[2]) < s);
}

__host__ __device__ void Vec3::MakeUnitVector()
{
    float k = 1.0 / sqrt(e[0] * e[0] + e[1] * e[1] + e[2] * e[2]);
    e[0] *= k; e[1] *= k; e[2] *= k;
}
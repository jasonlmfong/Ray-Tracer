#include "hip/hip_runtime.h"
#include "Camera.h"

__device__ Camera::Camera(Point3 lookfrom, Point3 lookat, Vec3 vup, float vfov, float aspect, float aperture, float focus_dist, float _time0, float _time1)
{ 
    // vfov is top to bottom in degrees
    float theta = DegreesToRadians(vfov);
    float half_height = tan(theta / 2.0f);
    float half_width = aspect * half_height;

    w = UnitVector(lookfrom - lookat);
    u = UnitVector(Cross(vup, w));
    v = Cross(w, u);

    origin = lookfrom;
    horizontal = 2.0f * half_width * focus_dist * u;
    vertical = 2.0f * half_height * focus_dist * v;
    lower_left_corner = origin - half_width * focus_dist * u - half_height * focus_dist * v - focus_dist * w;

    lens_radius = aperture / 2.0f;
    time0 = _time0;
    time1 = _time1;
}

__device__ Ray Camera::GetRay(float s, float t, hiprandState* local_rand_state)
{
    Vec3 rd = lens_radius * RandomInUnitDisk(local_rand_state);
    Vec3 offset = u * rd.x() + v * rd.y();
    return Ray(origin + offset, lower_left_corner + s * horizontal + t * vertical - origin - offset, RandomDoubleBetween(time0, time1, local_rand_state));
}
